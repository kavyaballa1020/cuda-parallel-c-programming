#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<malloc.h>
#include<stdlib.h>

__global__ void addKernel(int* a, int* b, int* c)
{
	*c = *a + *b;
}
__global__ void SubKernel(int* a, int* b, int* d)
{
	*d = *a - *b;
}

int main(void )
	{
	int *a, *b, *c,*d;
	int* d_a, * d_b, * d_c,*d_d;
	
	a = (int*)malloc(sizeof(int));
	b = (int*)malloc(sizeof(int));
	c = (int*)malloc(sizeof(int));
	d = (int*)malloc(sizeof(int));

	hipMalloc((int **)&d_a, sizeof(int));
	hipMalloc((int **)&d_b, sizeof(int));
	hipMalloc((int **)&d_c, sizeof(int));
	hipMalloc((int**)&d_d, sizeof(int));

	
	printf("Values of two numbers\n");
	scanf("%d%d", a, b);

	hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int), hipMemcpyHostToDevice);

	addKernel << <1, 10 >> > (d_a, d_b, d_c);
	SubKernel << <1, 5 >> > (d_a, d_b, d_d);


	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(d, d_d, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Substraction : %d\n", *d);

	return 0;
	}
