#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void evenOrOdd(int num) {
    if (num % 2 == 0) {
        printf("%d is even\n", num);
    }
    else {
        printf("%d is odd\n", num);
    }
}

int main() {
    int num;
    printf("enter number");
    scanf("%d", &num);
    evenOrOdd << <1, 1 >> > (num);
    hipDeviceSynchronize();
    return 0;
}


