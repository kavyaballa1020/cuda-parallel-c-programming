#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<malloc.h>

__global__ void matadd(int* l, int* m, int* n)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int id = gridDim.x * y + x;
    n[id] = n[id]+l[id] * m[id];
}
int main()
{
    int a[2][2];
    int b[2][2];
    int c[2][2];
    int* d, * e, * f;
    int i, j,k;
    printf("\n Enter elements of first matrix of size 2 * 2\n");
    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            scanf("%d", &a[i][j]);
        }
    }
    printf("\n Enter elements of second matrix of size 2 * 2\n");

    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            scanf("%d", &b[i][j]);
        }
    }
    hipMalloc((void**)&d, 2 * 2 * sizeof(int));
    hipMalloc((void**)&e, 2 * 2 * sizeof(int));
    hipMalloc((void**)&f, 2 * 2 * sizeof(int));
    hipMemcpy(d, a, 2 * 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, 2 * 2 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(2, 2);
    /* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */
    
    matadd << <grid, 1 >> > (d, e, f);
    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            c[i][j] = 0;
            for (k = 0; k < 2; k++)
            {
               
                c[i][j] = c[i][j] + a[i][k] * b[k][j];
                
            }
        }
        printf("\n");
    }
    hipMemcpy(c, f, 2 * 2 * sizeof(int), hipMemcpyDeviceToHost);
    printf("\nSum of two matrices:\n ");
    for (i = 0; i < 2; i++)
    {
        for (j = 0; j < 2; j++)
        {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);
    return 0;
}
