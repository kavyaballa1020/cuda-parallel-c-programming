#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include<malloc.h>


#define N 20

__global__ void add(int* a, int* b, int* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid <= 5)
        c[tid] = a[tid] + b[tid];
    else if(tid>5 && tid<=10)
        c[tid] = a[tid] * b[tid];
    else if (tid > 10 && tid <= 15)
        c[tid] = a[tid] - b[tid];
    else
        c[tid] = a[tid] / b[tid];

 
}

int main()
{
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add << <(N + 255) / 256, 256 >> > (dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        
        if (i <= 5)
        {
            if (i == 0)
            {
                printf("ADDITION\n");
            }
            printf("%d + %d = %d\n", a[i], b[i], c[i]);
        }
        else if(i>5 && i<=10)
        {
            if (i == 6)
            {
                printf("MULTIPLICATION\n");
            }
            printf("%d * %d = %d\n", a[i], b[i], c[i]);
        }
        else if (i > 10 && i <= 15)
        {
            if (i == 11)
            {
                printf("SUBSTRACTION\n");
            }
            printf("%d - %d = %d\n", a[i], b[i], c[i]);
        } 
        else
        {
            if (i == 16)
            {
                printf("DIVISION\n");
            }
            printf("%d / %d = %d\n", a[i], b[i], c[i]);
        }
            
        
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
